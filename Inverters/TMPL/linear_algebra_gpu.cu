#include "hip/hip_runtime.h"
#ifndef LINEAR_ALGEBRA_GPU_CU
#define LINEAR_ALGEBRA_GPU_CU

#include "global.h"
#include "gpu.h"



//reduction of the last elements
template<typename REAL>
__device__ void warpReduce(volatile REAL* sdata, int tid){
  sdata[tid]+=sdata[tid+32];
  sdata[tid]+=sdata[tid+16];
  sdata[tid]+=sdata[tid+8];
  sdata[tid]+=sdata[tid+4];
  sdata[tid]+=sdata[tid+2];
  sdata[tid]+=sdata[tid+1];
}
 //OLD VERSION
//Global sum calculation
template<typename REAL>
__global__ void global_sum_gpu(REAL* in, double* out, int N){
  __shared__ double sdata[GSUM_BLOCK_SIZE];
  int tid = threadIdx.x;
  int i;
  //  REAL res;
  sdata[tid]=in[tid];
  for (i=tid+GSUM_BLOCK_SIZE;i<N;i+= GSUM_BLOCK_SIZE){ // Sum over all blocks 
    sdata[tid]+=(double)in[i];
  }
  __syncthreads();
  
  for (i = GSUM_BLOCK_SIZE/2;i>32;i/=2){
    if (tid < i) sdata[tid]+=sdata[tid+i];
    __syncthreads();
  }
  if (tid<32){ //Unroll all the threads in a WARP
    warpReduce(sdata,tid);
  }
  __syncthreads();
  if (tid == 0)  out[0] = sdata[0];
}

////////////////////////////////////////////////////// Optimized Global Sum Kernel
// Requires multiple kernel calls in order to produce global synchronization 
// (You might think that this gives some overhead, but you are wrong.)
template<unsigned int blockSize, typename REAL, typename REALOUT>
__global__ void global_sum_gpu_opt(REAL * g_idata, REALOUT * g_odata, unsigned int n) {   

          __shared__ REALOUT sdata[blockSize];
	unsigned int tid = threadIdx.x; 
	unsigned int i = blockIdx.x*(blockSize*2) + tid; 
	unsigned int gridSize = blockSize*2*gridDim.x; sdata[tid] = 0;
	
	while (i < n) 
		{ 
		sdata[tid] += g_idata[i] + g_idata[i+blockSize]; 
		i += gridSize;						// gridSize loop maintains coalescing
		} 
	__syncthreads();
	
	// For Tesla cards we need not consider blockSize higher than 1024
	if (blockSize >= 1024) {				// Evaluated at compile time
		if (tid < 512) { 
			sdata[tid] += sdata[tid + 512]; 
			} 
		__syncthreads(); 
		} 
	if (blockSize >= 512) {				// Evaluated at compile time
		if (tid < 256) { 
			sdata[tid] += sdata[tid + 256]; 
			} 
		__syncthreads(); 
		} 
	if (blockSize >= 256) {				// Evaluated at compile time 
		if (tid < 128) { 
			sdata[tid] += sdata[tid + 128]; 
			} 
		__syncthreads(); 
		} 
	if (blockSize >= 128) { 				// Evaluated at compile time
		if (tid <	64) { 
			sdata[tid] += sdata[tid +	64]; 
			}
		__syncthreads(); 
		}
	if (tid < 32) warpReduce(sdata, tid); 	// Total unrolling and no sync in last WARP 
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	
}

template<unsigned int blockSize, typename REAL>
__global__ void global_sum_gpu_remainder(REAL * in, unsigned int n) {   
  __shared__ volatile REAL sdata[2*blockSize];
  unsigned int tid = threadIdx.x;

  sdata[tid]= (tid < n) ? in[tid] : 0;
  sdata[blockSize+tid]= 0;
  
    if (blockSize >=32) {sdata[tid]+=sdata[tid+16];}
    if (blockSize >=16) {sdata[tid]+=sdata[tid+8];}
    if (blockSize >=8) {sdata[tid]+=sdata[tid+4];}
    if (blockSize >=4) {sdata[tid]+=sdata[tid+2];}
    if (blockSize >=2) {sdata[tid]+=sdata[tid+1];}
  
  
  __syncthreads();
  if (tid == 0)  in[0] = sdata[0];
}




//Reduction of the last elements
template<typename COMPLEX>
__device__ void warpReduce_complex(volatile COMPLEX* sdata, int tid){
  _complex_add(sdata[tid],sdata[tid],sdata[tid+32]);
  _complex_add(sdata[tid],sdata[tid],sdata[tid+16]);
  _complex_add(sdata[tid],sdata[tid],sdata[tid+8]);
  _complex_add(sdata[tid],sdata[tid],sdata[tid+4]);
  _complex_add(sdata[tid],sdata[tid],sdata[tid+2]);
  _complex_add(sdata[tid],sdata[tid],sdata[tid+1]);
}

////////////////////////////////////////////////////// Optimized Global Sum Kernel
// Requires multiple kernel calls in order to produce global synchronization 
// (You might think that this gives some overhead, but you are wrong.)
template<unsigned int blockSize, typename COMPLEX>
__global__ void global_complex_sum_gpu_opt(COMPLEX * g_idata, complex * g_odata, unsigned int n) {   

          __shared__ complex sdata[blockSize];
	unsigned int tid = threadIdx.x; 
	unsigned int i = blockIdx.x*(blockSize*2) + tid; 
	unsigned int gridSize = blockSize*2*gridDim.x; sdata[tid].re = 0,sdata[tid].im = 0;
	
	while (i < n) 
		{ 
		sdata[tid].re += g_idata[i].re + g_idata[i+blockSize].re; 
		sdata[tid].im += g_idata[i].im + g_idata[i+blockSize].im; 
		i += gridSize;	
		} 
	__syncthreads();
	
	// For Tesla cards we need not consider blockSize higher than 1024
	if (blockSize >= 1024) {				// Evaluated at compile time
		if (tid < 512) { 
			sdata[tid].re += sdata[tid + 512].re; 
			sdata[tid].im += sdata[tid + 512].im; 
			} 
		__syncthreads(); 
		} 
	if (blockSize >= 512) {				// Evaluated at compile time
		if (tid < 256) { 
			sdata[tid].re += sdata[tid + 256].re; 
			sdata[tid].im += sdata[tid + 256].im; 
			} 
		__syncthreads(); 
		} 
	if (blockSize >= 256) {				// Evaluated at compile time 
		if (tid < 128) { 
			sdata[tid].re += sdata[tid + 128].re; 
			sdata[tid].im += sdata[tid + 128].im; 
			} 
		__syncthreads(); 
		} 
	if (blockSize >= 128) { 				// Evaluated at compile time
		if (tid <	64) { 
			sdata[tid].re += sdata[tid + 64].re; 
			sdata[tid].im += sdata[tid + 64].im; 
			}
		__syncthreads(); 
		}
	if (tid < 32) warpReduce_complex(sdata, tid); 	// Total unrolling and no sync in last WARP 
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
	
}
template<unsigned int blockSize, typename COMPLEX>
__global__ void global_complex_sum_gpu_remainder(COMPLEX * in, unsigned int n) {   
  __shared__ volatile COMPLEX sdata[2*blockSize];
  unsigned int tid = threadIdx.x;

  if (tid<n) {sdata[tid].re=in[tid].re;sdata[tid].im=in[tid].im;}
  else{ sdata[tid].re=0;sdata[tid].im=0;}
  sdata[blockSize+tid].re= 0;
  sdata[blockSize+tid].re= 0;
  
    if (blockSize >=32) {_complex_add(sdata[tid],sdata[tid],sdata[tid+16]);}
    if (blockSize >=16) {_complex_add(sdata[tid],sdata[tid],sdata[tid+8]);}
    if (blockSize >=8) {_complex_add(sdata[tid],sdata[tid],sdata[tid+4]);}
    if (blockSize >=4) {_complex_add(sdata[tid],sdata[tid],sdata[tid+2]);}
    if (blockSize >=2) {_complex_add(sdata[tid],sdata[tid],sdata[tid+1]);}
  
  
  __syncthreads();
  if (tid == 0) {in[0].re = sdata[0].re;in[0].im = sdata[0].im;};
}




//Global sum calculation  - OLD VERSION
template<typename COMPLEX>
__global__ void global_sum_complex_gpu(COMPLEX* in, complex* out, int N){
  __shared__ complex sdata[GSUM_BLOCK_SIZE];
  int tid = threadIdx.x;
  int i;
  //  REAL res;
  sdata[tid].re=in[tid].re;
  sdata[tid].im=in[tid].im;
  for (i=tid+GSUM_BLOCK_SIZE;i<N;i+= GSUM_BLOCK_SIZE){ // Sum over all blocks 
    _complex_add(sdata[tid],sdata[tid],in[i]);    
  }
  __syncthreads();
  
  for (i = GSUM_BLOCK_SIZE/2;i>32;i/=2){
    if (tid < i){
      _complex_add(sdata[tid],sdata[tid],sdata[tid+i]);
    }
    __syncthreads();
  }
  if (tid<32){ //Unroll all the threads in a WARP
    warpReduce_complex(sdata,tid);
  }
  __syncthreads();
  if (tid == 0)  out[0] = sdata[0];
}

/* Re <s1,s2>  FOR OPTIMIZED GLOBAL SUM*/
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_prod_re_padded_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,unsigned int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
//  i=min(i,N-1);
  resField[i] = (i<N)? _complex_prod_re(s1[i],s2[i]) : 0;
}
/* Im <s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_prod_im_padded_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,unsigned int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  //i=min(i,N-1);
  resField[i]= (i<N) ? _complex_prod_im(s1[i],s2[i]) : 0;
}

/* <s1,s2> */
template< typename COMPLEX>
  __global__ void spinor_field_prod_padded_gpu(COMPLEX* s1, COMPLEX* s2, complex* resField,unsigned int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
//  i=min(i,N-1);
  if (i<N) {_complex_prod(resField[i],s1[i],s2[i]);}
  else{ resField[i].re=0;resField[i].im=0;}
}

/* Re <g5*s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_g5_prod_re_padded_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,unsigned int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
 // i=min(i,N-1);
  resField[i]=(i<N)? _complex_prod_re(s1[i],s2[i]) :0;
  if (i>((N>>1)-1)){
  	resField[i]=-resField[i];
  }
}

/* Im <g5*s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_g5_prod_im_padded_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,unsigned int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
//  i=min(i,N-1);
  resField[i]=(i<N)? _complex_prod_im(s1[i],s2[i]) :0;
  if (i>((N>>1)-1)){
  	resField[i]=-resField[i];
  }
}

/* Re <s1,s1> */ 
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_sqnorm_padded_gpu(COMPLEX* s1, REAL* resField,unsigned int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
//  i=min(i,N-1);
  resField[i] =(i<N)?  _complex_prod_re(s1[i],s1[i]) : 0;
}

/////////// *********************



/* Re <s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_prod_re_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  resField[i] = _complex_prod_re(s1[i],s2[i]);
}

/* Im <s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_prod_im_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  resField[i]=_complex_prod_im(s1[i],s2[i]);
}

/* <s1,s2> */
template< typename COMPLEX>
  __global__ void spinor_field_prod_gpu(COMPLEX* s1, COMPLEX* s2, COMPLEX* resField,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_prod(resField[i],s1[i],s2[i]);
}

/* Re <g5*s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_g5_prod_re_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  resField[i]=_complex_prod_re(s1[i],s2[i]);
  if (i>((N>>1)-1)){
  	resField[i]=-resField[i];
  }
}

/* Im <g5*s1,s2> */
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_g5_prod_im_gpu(COMPLEX* s1, COMPLEX* s2, REAL* resField,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  resField[i]=_complex_prod_im(s1[i],s2[i]);
  if (i>((N>>1)-1)){
  	resField[i]=-resField[i];
  }
}

/* Re <s1,s1> */ 
template<typename COMPLEX, typename REAL>
  __global__ void spinor_field_sqnorm_gpu(COMPLEX* s1, REAL* resField,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  resField[i] = _complex_prod_re(s1[i],s1[i]);
}

/* s1+=r*s2 r real */
template< typename COMPLEX, typename REAL >
__global__ void spinor_field_mul_add_assign_gpu(COMPLEX *s1, REAL r, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_mulr_assign(s1[i],r,s2[i]);
}

/* s1=r*s2 r real */
template< typename COMPLEX , typename REAL >
__global__ void spinor_field_mul_gpu(COMPLEX *s1, REAL r, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_mulr(s1[i],r,s2[i]);
}

/* s1+=c*s2 c complex */
template< typename COMPLEX >
__global__ void spinor_field_mulc_add_assign_gpu(COMPLEX *s1, COMPLEX c, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_mul_assign(s1[i],c,s2[i]);
}

/* s1=c*s2 c complex */
template< typename COMPLEX >
__global__ void spinor_field_mulc_gpu(COMPLEX *s1, COMPLEX c, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_mul(s1[i],c,s2[i]);
}

/* r=s1+s2 */
template< typename COMPLEX>
__global__ void spinor_field_add_gpu(COMPLEX *r, COMPLEX *s1, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_add(r[i],s1[i],s2[i]);

}

/* r=s1-s2 */
template< typename COMPLEX >
__global__ void spinor_field_sub_gpu(COMPLEX *r, COMPLEX * s1, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_sub(r[i],s1[i],s2[i]);
}

/* s1+=s2 */
template< typename COMPLEX>
__global__ void spinor_field_add_assign_gpu(COMPLEX *s1, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_add_assign(s1[i],s2[i]);
}

/* s1-=s2 */
template< typename COMPLEX >
__global__ void spinor_field_sub_assign_gpu(COMPLEX* s1, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_sub_assign(s1[i],s2[i]);
}

/* s1=0 */
template< typename COMPLEX>
__global__ void spinor_field_zero_gpu(COMPLEX *s1,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_0(s1[i]);
}

/* s1=-s2 */
template< typename COMPLEX >
__global__ void spinor_field_minus_gpu(COMPLEX* s1, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_minus(s1[i],s2[i]);
}

/* s1=r1*s2+r2*s3 */
template< typename COMPLEX , typename REAL >
__global__ void spinor_field_lc_gpu(COMPLEX *s1, REAL r1, COMPLEX *s2, REAL r2, COMPLEX *s3, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_rlc(s1[i],r1,s2[i],r2,s3[i]);
}

/* s1+=r*s2 r real */
template< typename COMPLEX, typename REAL >
__global__ void spinor_field_lc_add_assign_gpu(COMPLEX *s1, REAL r1, COMPLEX *s2, REAL r2, COMPLEX *s3, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_rlc_assign(s1[i],r1,s2[i],r2,s3[i]);
}


/* s1=cd1*s2+cd2*s3 cd1, cd2 complex*/
template< typename COMPLEX >
__global__ void spinor_field_clc_gpu(COMPLEX *s1, COMPLEX c1, COMPLEX *s2, COMPLEX c2, COMPLEX *s3, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_clc(s1[i],c1,s2[i],c2,s3[i]);
}

/* s1+=r*s2 r real */
template< typename COMPLEX >
__global__ void spinor_field_clc_add_assign_gpu(COMPLEX *s1, COMPLEX c1, COMPLEX *s2, COMPLEX c2, COMPLEX *s3, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_clc_assign(s1[i],c1,s2[i],c2,s3[i]);
}

/* s1=g5*s2  */
template< typename COMPLEX>
__global__ void spinor_field_g5_gpu(COMPLEX *s1, COMPLEX *s2,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  if ( i < (N>>1) ) {
    s1[i]=s2[i];
  }
  else{
    _complex_minus(s1[i],s2[i]);
  }
}

/* s1=g5*s1 */
template< typename COMPLEX >
__global__ void spinor_field_g5_assign_gpu(COMPLEX* s1,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  if (i>((N>>1)-1)){
    _complex_minus(s1[i],s1[i]);
  }
}

/* tools per eva.c  */
template< typename COMPLEX , typename REAL >
__global__ void spinor_field_lc1_gpu(REAL r, COMPLEX *s1, COMPLEX *s2, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_mulr_assign(s1[i],r,s2[i]);
}


template< typename COMPLEX, typename REAL >
__global__ void spinor_field_lc2_gpu(REAL r1, REAL r2, COMPLEX *s1, COMPLEX *s2, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_rlc(s1[i],r1,s1[i],r2,s2[i]);
}

template< typename COMPLEX , typename REAL >
__global__ void spinor_field_lc3_gpu(REAL r1,REAL r2, COMPLEX *s1, COMPLEX *s2, COMPLEX *s3, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  _complex_rlc_assign(s3[i],r1,s1[i],r2,s2[i]);
  _complex_minus(s3[i],s3[i]);
}

template< typename COMPLEX>
__global__ void spinor_field_copy_gpu_to_gpu_gpu(COMPLEX* dst, COMPLEX* src, int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  i=min(i,N-1);
  dst[i]=src[i];
}

/* c1=0 */
/*
template< typename COMPLEX>
__global__ void complex_field_zero_gpu(COMPLEX *c1,int N){
  int i = blockIdx.x*BLOCK_SIZE + threadIdx.x;
  if (i<N) {
    c1[i].re=0;
    c1[i].im=0;
  }
}
*/


#endif
